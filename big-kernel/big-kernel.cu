#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// std::system includes
#include <cstdio>

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

// Semaphore include
#include <sys/types.h>
#include <unistd.h>
#include "sysvbarrier.h"

// throw error on equality
#define ERR_EQ(X,Y) do { if ((X) == (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

// throw error on difference
#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

#define ROUND_UP(N, BASE) \
  (N + BASE - 1) / BASE

// copy from source -> destination arrays
__device__ void slow_kernel(int *dst, int *src, int n, int delay) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  for (volatile int i = 0; i < delay; i++);
  if (id < n) {
    dst[id] = src[id];
  }
}

// Named kernels for easier profiling
__global__ void low_priority(int *dst, int *src, int n, int delay) {
  slow_kernel(dst, src, n, delay);
}

__global__ void high_priority(int *dst, int *src, int n, int delay) {
  slow_kernel(dst, src, n, delay);
}

// initialize memory
void mem_init(int *buf, size_t n) {
  for (int i = 0; i < n; i++) {
    buf[i] = i;
  }
}

// Forward declarations
hipError_t setup_memory(int* src[], int* dst[], size_t size, size_t n_regions);
void run_experiment(const int priority, const int size, const int iterations,
    const int delay);

int main(int argc, char **argv) {
  hipDeviceProp_t device_prop;
  int dev_id;

  printf("Starting [%s]...\n", argv[0]);

  // set device
  dev_id = findCudaDevice(argc, (const char **) argv);
  checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
  if ((device_prop.major << 4) + device_prop.minor < 0x35) {
    fprintf(stderr,
        "%s requires Compute Capability of SM 3.5 or higher to run.\nexiting...\n",
        argv[0]);
    exit (EXIT_WAIVED);
  }

  // command line args
  const int delay = getCmdLineArgumentInt(argc, (const char **) argv, "delay");
  const size_t size = getCmdLineArgumentInt(argc, (const char **) argv, "size");
  const int priority = getCmdLineArgumentInt(argc, (const char **) argv, "priority");
  const int iterations = getCmdLineArgumentInt(argc, (const char **) argv, "iterations");
  const int n_processes = getCmdLineArgumentInt(argc, (const char **) argv, "processes");

  // get the range of priorities available
  // [ greatest_priority, least_priority ]
  int priority_low;
  int priority_hi;
  checkCudaErrors(hipDeviceGetStreamPriorityRange(&priority_low,
      &priority_hi));

  if (size == 0 || iterations == 0) {
    fprintf(stderr,
        "Please provide --size=<int> --priority=<int> --iterations=<int> "
        "--processes=<int> and --delay=<int> (optional) flags.\nexting...\n");
    exit (EXIT_FAILURE);
  } else {
    fprintf(stdout, "Called with arguments size %zu, priority %d, iterations %d, n_processes %d, and delay %d\n",
        size, priority, iterations, n_processes, delay);
  }
  
  if (priority_hi > priority || priority_low < priority) {
    fprintf(stderr, "Priority must be within %d and %d.\nexting...\n",
        priority_hi, priority_low);
    exit (EXIT_FAILURE);
  }

  // Set up barriers
#define BARRIER_ID_START 149
  BARRIER start_barrier = barrier_create(BARRIER_ID_START, n_processes);
  pid_t id = 2;//getpid();

  // Set kernel to run
  void (*kernel)(int*, int*, int, int) = priority ? &high_priority : &low_priority;
  
  // Create memory regions
#define N_MEMORY_REGIONS 8
  size_t n_regions = max(min(N_MEMORY_REGIONS, iterations / 4), 1);
  int *src[n_regions];
  int *dst[n_regions];
  
  setup_memory(src, dst, size, n_regions);
  hipDeviceSynchronize();

  // Create stream
  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithPriority(&stream,
      hipStreamNonBlocking, priority));
 
  // Compute number of threads and blocks
  int blockSize;
  int minGridSize;
  int gridSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel);
  gridSize = (size + blockSize - 1) / blockSize;
 
  // launch the kernel iteration times.
  // each consecutive launch uses a different memory region
  for (int i = 0; i < ROUND_UP(iterations, n_regions); i++) {
    for (int j = 0; j < n_regions; j++) {
      printf("waiting at barrier 0\n");
      barrier_wait(start_barrier, id, n_processes);
      printf("exited barrier 0\n");
      kernel<<<gridSize, blockSize, 0, stream>>>(dst[j], src[j], size,
          delay);
      checkCudaErrors(hipStreamSynchronize(stream));
    }
  }

  // check results of the last computation
  for (int i = 0; i < n_regions; i++) {
    ERR_NE(memcmp(dst[i], src[i], size), 0);  
  }
  
  // Clean up
  for (int i = 0; i < n_regions; i++) {
    checkCudaErrors (hipFree(src[i]));
    checkCudaErrors(hipFree(dst[i]));
  }
  exit (EXIT_SUCCESS);
}

hipError_t setup_memory(int* src[], int* dst[], size_t size,
    size_t n_regions) {
  for (int i = 0; i < n_regions; i++) {
    checkCudaErrors(hipMallocManaged(&src[i], size * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&dst[i], size * sizeof(int)));
    mem_init(src[i], size);
    memset(dst[i], 0, size);
  }
  return hipSuccess;
}

